
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void csr_mv(int m, double *AVal, int *ARowP, int *AColI, double *x, double *b)
{
	int i = ((blockDim.x * blockIdx.x) + threadIdx.x), j;
	if (i < m) {
		for (j = ARowP[i]; j < ARowP[i+1]; j++)
		{
			b[i] += AVal[j]*x[AColI[j]];
		}}
}

int main()
{
	// Define the Sparse Matrix
	int n0 = 80000;   // No. of no. zero enteries
	int m = 8000;     // m rows n collumns for A
	int n = m;
	int ARowP[m+1];
	int AColI[n0];
	double AVal[n0];

	double x[n];	// b=Ax
	double b[m];
	int thrd = 1000;
	int bloc = m/1000;

	int i,j;
//	Value Assignment
	
	for (i=0;i<n0;i++){
		AVal[i] = 1+i;
		AColI[i] = rand() % m;
        }

	for (i=0;i<(m+1);i++)
		ARowP[i] = (i-1)*10;

	for (i=0;i<n;i++)
		x[i] = 1;

	for (i=0;i<m;i++)
		b[i] = 0;


//  Data comminication

	double *d_AVal, *d_x, *d_b;
	int *d_AColI, *d_ARowP;

	hipMalloc(&d_AVal, n0*sizeof(double));
	hipMalloc(&d_x, n*sizeof(double));
	hipMalloc(&d_b, m*sizeof(double));
	hipMalloc(&d_ARowP, (m+1)*sizeof(int));
	hipMalloc(&d_AColI, n0*sizeof(int));

	hipMemcpy(d_AVal, AVal, n0*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, m*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_ARowP, ARowP, (m+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_AColI, AColI, n0*sizeof(int), hipMemcpyHostToDevice);

	csr_mv <<< bloc, thrd >>> ( m, d_AVal, d_ARowP, d_AColI, d_x, d_b);

	hipMemcpy(b, d_b, m*sizeof(double), hipMemcpyDeviceToHost);

	for (i = 0; i < 100; i++)
		printf("b[%d] = %lf\n", i, b[i]);

	return 0;
}
