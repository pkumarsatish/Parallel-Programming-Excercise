#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

///////////////////////////////////////////////////////////////////////////////
//////Coalesced Kernal Function ///////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
 __global__ void Ax1(float *d_A, float *d_x, float *d_b, int *d_rs, int *d_NumBlocks,int *d_NumThreads){

	int ThreadId = threadIdx.x;
	int BlockId = blockIdx.x;
	int i, cs;

	cs = ThreadId + BlockId*(*d_NumBlocks);
	for(i=0; i<(*d_rs); i++)
		d_b[i*(*d_rs)+cs] = d_A[i*(*d_rs)+cs]*d_x[cs];
}

///////////////////////////////////////////////////////////////////////////////
//////Non-Coalesced Kernal Function ///////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
 __global__ void Ax2(float *d_A, float *d_x, float *d_b, int *d_rs, int *d_NumBlocks,int *d_NumThreads){

	int ThreadId = threadIdx.x;
	int BlockId = blockIdx.x;
	int i, cs; 
	
	cs = ThreadId + BlockId*(*d_NumBlocks);
    	for(i=0; i<(*d_rs); i++)
		d_b[i+cs*(*d_rs)] = d_A[cs*(*d_rs)+i]*d_x[i]
}


int main( int argc, char** argv)
{	
	int NumRow = 12800, NumCol = 12800;
        float A[NumRow][NumCol], x[NumCol], b[NumRow];
    
	int NumThreads = 256, NumBlocks = 50;
	int rs,cs;
	
	int i,j; 	
	struct timespec ti, tf;
	float dt;	
	
	for(i=0;i<NumRow;i++)				// Assigning values for A and x
		for(j=0;j<NumCol;j++)
			A[i][j] = 0.1/i+j;
	for(j=0;j<NumCol;j++)
		x[j] = 0.1/j;


   	float *d_A, *d_x,*d_b;				// Device Variables
	int *d_NumThreads,*d_NumBlocks;
	int *d_rs,*d_cs;

	hipMalloc((void**)&d_A, sizeof(float)*NumRow*NumCol);
	hipMalloc((void**)&d_x, sizeof(float)*NumCol);
	hipMalloc((void**)&d_b, sizeof(float)*NumRow*NumCol);
        hipMalloc((void**)&d_cs, sizeof(int));
        hipMalloc((void**)&d_rs, sizeof(int));
	hipMalloc((void**)&d_NumThreads, sizeof(int));
        hipMalloc((void**)&d_NumBlocks, sizeof(int));

	hipMemcpy(d_A, A, sizeof(float)*NumRow*NumCol , hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, sizeof(float)*NumCol, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(float)*NumRow*NumCol, hipMemcpyHostToDevice);
        hipMemcpy(d_cs, &rs, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_rs, &cs, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_NumThreads, &NumThreads, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_NumBlocks, &NumBlocks, sizeof(int), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

        clock_gettime(CLOCK_MONOTONIC,&ti);
	Ax1<<<50,256>>>( d_A,d_x,d_b, d_rs,d_NumBlocks, d_NumThreads);	// Coalesced + Wraped
	hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &tf);
        dt = (float)((tf.tv_sec-ti.tv_sec)*1e9 + tf.tv_nsec - ti.tv_nsec);
	printf("Coaleasced + Wraped: %lf \n", dt);
	
        clock_gettime(CLOCK_MONOTONIC,&ti);
	Ax1<<<128,100>>>( d_A,d_x,d_b, d_rs,d_NumBlocks, d_NumThreads);	// Coalesced + Non-Wraped
	hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &tf);
        dt = (float)((tf.tv_sec-ti.tv_sec)*1e9 + tf.tv_nsec - ti.tv_nsec);
	printf("Coa + Non-Wraped: %lf \n", dt);

	clock_gettime(CLOCK_MONOTONIC,&ti);
	Ax2<<<50,256>>>( d_A,d_x,d_b, d_rs,d_NumBlocks, d_NumThreads);	// Non-Coalesced + Wraped
	hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &tf);
        dt = (float)((tf.tv_sec-ti.tv_sec)*1e9 + tf.tv_nsec - ti.tv_nsec);
	printf("Non-Coa + Wraped: %lf \n", dt);

        clock_gettime(CLOCK_MONOTONIC,&ti);
	Ax2<<<128,100>>>( d_A,d_x,d_b, d_rs,d_NumBlocks, d_NumThreads);	// Non-Coalesced + Non-Wraped
	hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &tf);
        dt = (float)((tf.tv_sec-ti.tv_sec)*1e9 + tf.tv_nsec - ti.tv_nsec);
	printf("Non-Coa + Non-Wraped: %lf \n", dt);

	hipMemcpy(b, d_b, sizeof(float)*NumRow*NumCol, hipMemcpyDeviceToHost);    
        hipDeviceSynchronize();
    
	return 0;
}
